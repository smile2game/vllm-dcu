#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <algorithm>

#include "attention_dtypes.h"
#include "attention_utils.cuh"

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
  #include "../quantization/fp8/amd/quant_utils.cuh"
typedef __hip_bfloat16 __hip_bfloat16;
#else
  #include "../quantization/fp8/nvidia/quant_utils.cuh"
#endif

#ifndef USE_ROCM
  #define WARP_SIZE 32
#else
  #define WARP_SIZE warpSize
#endif


#include "static_switch.h"
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace vllm {

// Utility function for attention softmax.
template <int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  // Decompose the thread index into warp / lane.
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

  // Compute the sum per warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Warp leaders store the data to shared memory.
  if (lane == 0) {
    red_smem[warp] = sum;
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The warps compute the final sums.
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }

  // Parallel reduction inside the warp.
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }

  // Broadcast to other threads.
  return VLLM_SHFL_SYNC(sum, 0);
}

// TODO(woosuk): Merge the last two dimensions of the grid.
// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int REUSE_KV_TIMES = 1,
          bool odd_nheads = false,
          int PARTITION_SIZE = 0,std::enable_if_t<!std::is_same<scalar_t, uint16_t>::value, int> = 0>  // Zero means no partitioning.
__device__ void paged_attention_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_heads,                   // [num_heads]
    const int num_kv_heads,               // [num_kv_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {}

// TODO(woosuk): Merge the last two dimensions of the grid.
// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int REUSE_KV_TIMES = 1,
          bool odd_nheads = false,
          int PARTITION_SIZE = 0,std::enable_if_t<std::is_same<scalar_t, uint16_t>::value, int> = 0>  // Zero means no partitioning.
__device__ void paged_attention_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                     // max_num_partitions]
    scalar_t* __restrict__ out,  // [num_seqs, num_heads, max_num_partitions,
                                 // head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_heads,                   // [num_heads]
    const int num_kv_heads,               // [num_kv_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  const int seq_idx = blockIdx.z;
  const int partition_idx = blockIdx.y;
  const int max_num_partitions = gridDim.y;
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int seq_len = seq_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= seq_len) {
    // No work to do. Terminate the thread block.
    return;
  }

  const int num_seq_blocks = DIVIDE_ROUND_UP(seq_len, BLOCK_SIZE);
  const int num_blocks_per_partition =
      USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_seq_blocks;
  const int partition_size = USE_PARTITIONING ? PARTITION_SIZE : num_seq_blocks * BLOCK_SIZE;
  // [start_block_idx, end_block_idx) is the range of blocks to process.
  const int start_block_idx =
      USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx =
      MIN(start_block_idx + num_blocks_per_partition, num_seq_blocks);
  const int num_blocks = end_block_idx - start_block_idx;

  // [start_token_idx, end_token_idx) is the range of tokens to process.
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx =
      MIN(start_token_idx + num_blocks * BLOCK_SIZE, seq_len);
  const int num_tokens = end_token_idx - start_token_idx;

  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_THREAD_GROUPS =
      NUM_THREADS / THREAD_GROUP_SIZE;  // Note: This assumes THREAD_GROUP_SIZE
                                        // divides NUM_THREADS
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP =
      DIVIDE_ROUND_UP(BLOCK_SIZE, WARP_SIZE);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  // const int warp_idx_vec = thread_idx / WARP_SIZE;
  // int warp_idx =0;
  // asm volatile("v_readfirstlane_b32 %0,%1"
  //               : "=s"(warp_idx)
  //               : "v"(warp_idx_vec)
  //               :);
  // // const int warp_idx = thread_idx / WARP_SIZE;

  // const int lane = thread_idx % WARP_SIZE;

    //const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;

  int warp_id_vec = threadIdx.x / WARP_SIZE; //warp id in a block
  int warp_idx =0;
  asm volatile("v_readfirstlane_b32 %0,%1"
                : "=s"(warp_idx)
                : "v"(warp_id_vec)
                :);

  // const int head_idx = blockIdx.x;
  // const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  // const float alibi_slope =
  //     alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];

  // A vector type to store a part of a key or a query.
  // The vector size is configured in such a way that the threads in a thread
  // group fetch or compute 16 bytes at a time. For example, if the size of a
  // thread group is 4 and the data type is half, then the vector size is 16 /
  // (4 * sizeof(half)) == 2.
  constexpr int VEC_SIZE = MAX(32 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Quant_vec = typename Vec<cache_t, VEC_SIZE>::Type;

  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;

  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;

  // Load the query to registers.
  // Each thread in a thread group has a different part of the query.
  // For example, if the the thread group size is 4, then the first thread in
  // the group has 0, 4, 8, ... th vectors of the query, and the second thread
  // has 1, 5, 9, ... th vectors of the query, and so on. NOTE(woosuk): Because
  // q is split from a qkv tensor, it may not be contiguous.
  // const scalar_t* q_ptr = q + seq_idx * q_stride;
  const scalar_t* q_ptr_offset = q + seq_idx * q_stride;

  __shared__ Q_vec q_vecs[REUSE_KV_TIMES * THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
// #pragma unroll
//   for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD;
//        i += NUM_THREAD_GROUPS) {
//     const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
//     q_vecs[thread_group_offset][i] =
//         *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
//   }
//   __syncthreads();  // TODO(naed90): possible speedup if this is replaced with a
//                     // memory wall right before we use q_vecs

  // Memory planning.
  extern __shared__ char shared_mem[];
  // NOTE(woosuk): We use FP32 for the softmax logits for better accuracy.
  float* logits = reinterpret_cast<float*>(shared_mem);
  // Workspace for reduction.
  __shared__ float red_smem[REUSE_KV_TIMES][2 * NUM_WARPS];
  // float (*red_smem)[2 * NUM_WARPS] = reinterpret_cast<float(*)[2 * NUM_WARPS]>(&shared_mem[10*1024]);

  // __shared__ char shared_mem[12 * 1024];
  // float* logits = reinterpret_cast<float*>(shared_mem);
  // __shared__ float red_smem[REUSE_KV_TIMES][2 * NUM_WARPS];

  // x == THREAD_GROUP_SIZE * VEC_SIZE
  // Each thread group fetches x elements from the key at a time.
  constexpr int x = 16 / sizeof(cache_t);
  float qk_max[REUSE_KV_TIMES];

  for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
      qk_max[reuse_kv_idx] = -FLT_MAX;
  }
   
  const int num_blocks_per_kv = ((num_queries_per_kv + REUSE_KV_TIMES -1) / REUSE_KV_TIMES);
  const int head_idx_soffset = (blockIdx.x / num_blocks_per_kv) * num_queries_per_kv + (blockIdx.x % num_blocks_per_kv) * REUSE_KV_TIMES;
  const int kv_head_idx = head_idx_soffset / num_queries_per_kv;
  const int q_boundary = (kv_head_idx + 1)* num_queries_per_kv;

  #pragma unroll
  for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
    const int head_idx = head_idx_soffset + reuse_kv_idx;//blockIdx.x * REUSE_KV_TIMES + reuse_kv_idx;
    const scalar_t* q_ptr = q_ptr_offset + head_idx * HEAD_SIZE;
    #pragma unroll
    for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD; i += NUM_THREAD_GROUPS) {
      const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
      q_vecs[reuse_kv_idx*THREAD_GROUP_SIZE + thread_group_offset][i] = *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
    }
  }
  __syncthreads(); // TODO(naed90): possible speedup if this is replaced with a memory wall right before we use q_vecs

  // Iterate over the key blocks.
  // Each warp fetches a block of keys for each iteration.
  // Each thread group in a warp fetches a key from the block, and computes
  // dot product with the query.
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx; block_idx += NUM_WARPS) {
    // NOTE(woosuk): The block number is stored in int32. However, we cast it to
    // int64 because int32 can lead to overflow when this variable is multiplied
    // by large numbers (e.g., kv_block_stride).
    // For blocksparse attention: skip computation on blocks that are not
    // attended
    for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
    const int head_idx = head_idx_soffset + reuse_kv_idx;//blockIdx.x * REUSE_KV_TIMES + reuse_kv_idx;
    if(!odd_nheads || head_idx < q_boundary) {
        // blocksparse specific vars
    int bs_block_offset;
    int q_bs_block_id;
    if constexpr (IS_BLOCK_SPARSE) { 
      // const int num_blocksparse_blocks = DIVIDE_ROUND_UP(seq_len,
      // blocksparse_block_size);
      q_bs_block_id = (seq_len - 1) / blocksparse_block_size;
      if (blocksparse_head_sliding_step >= 0)
        // sliding on q heads
        bs_block_offset =
            (tp_rank * num_heads + head_idx) * blocksparse_head_sliding_step + 1;
      else
        // sliding on kv heads
        bs_block_offset = (tp_rank * num_kv_heads + kv_head_idx) *
                              (-blocksparse_head_sliding_step) +
                          1;
    }
    if constexpr (IS_BLOCK_SPARSE) {
      const int k_bs_block_id = block_idx * BLOCK_SIZE / blocksparse_block_size;
      const bool is_remote =
          ((k_bs_block_id + bs_block_offset) % blocksparse_vert_stride == 0);
      const bool is_local =
          (k_bs_block_id > q_bs_block_id - blocksparse_local_blocks);
      if (!is_remote && !is_local) {
        for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
          const int physical_block_offset =
              (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
          const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;

          if (thread_group_offset == 0) {
            // NOTE(linxihui): assign very large number to skipped tokens to
            // avoid contribution to the sumexp softmax normalizer. This will
            // not be used at computing sum(softmax*v) as the blocks will be
            // skipped.
            logits[token_idx - start_token_idx] = -FLT_MAX;
          }
        }
        continue;
      }
    }
    const float alibi_slope = alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];
    const int64_t physical_block_number = static_cast<int64_t>(block_table[block_idx]);

    // Load a key to registers.
    // Each thread in a thread group has a different part of the key.
    // For example, if the the thread group size is 4, then the first thread in
    // the group has 0, 4, 8, ... th vectors of the key, and the second thread
    // has 1, 5, 9, ... th vectors of the key, and so on.
    for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset = (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      K_vec k_vecs[NUM_VECS_PER_THREAD];
      if(reuse_kv_idx == 0) {
        #pragma unroll
        for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
          const cache_t* k_ptr =
              k_cache + physical_block_number * kv_block_stride +
              kv_head_idx * kv_head_stride + physical_block_offset * x;
          const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
          const int offset1 = (vec_idx * VEC_SIZE) / x;
          const int offset2 = (vec_idx * VEC_SIZE) % x;

          if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
            k_vecs[j] = *reinterpret_cast<const K_vec*>(
                k_ptr + offset1 * BLOCK_SIZE * x + offset2);
          } else {
            // Vector conversion from Quant_vec to K_vec.
            Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(
                k_ptr + offset1 * BLOCK_SIZE * x + offset2);
            k_vecs[j] = fp8::scaled_convert<K_vec, Quant_vec, KV_DTYPE>(
                k_vec_quant, kv_scale);
          }
        }
      }
      __builtin_amdgcn_sched_barrier(0);
      // Compute dot product.
      // This includes a reduction across the threads in the same thread group.
      float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(q_vecs[reuse_kv_idx*THREAD_GROUP_SIZE + thread_group_offset], k_vecs);
      // Add the ALiBi bias if slopes are given.
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - seq_len + 1) : 0;
      __builtin_amdgcn_sched_barrier(0);
      if (thread_group_offset == 0) {
        // Store the partial reductions to shared memory.
        // NOTE(woosuk): It is required to zero out the masked logits.
        const bool mask = token_idx >= seq_len;
        logits[(reuse_kv_idx * partition_size) + (token_idx - start_token_idx)] = mask ? 0.f : qk;
        // Update the max value.
        qk_max[reuse_kv_idx] = mask ? qk_max[reuse_kv_idx] : fmaxf(qk_max[reuse_kv_idx], qk);
      }
    }
  }
  }
  }
  // Get the sum of the exp values.
  float exp_sum[REUSE_KV_TIMES] = {0.f};

  // Perform reduction across the threads in the same warp to get the
  // max qk value for each "warp" (not across the thread block yet).
  // The 0-th thread of each thread group already has its max qk value.
  for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
    const int head_idx = head_idx_soffset + reuse_kv_idx;
    if(!odd_nheads || head_idx < q_boundary) {
      #pragma unroll
      for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
        qk_max[reuse_kv_idx] = fmaxf(qk_max[reuse_kv_idx], VLLM_SHFL_XOR_SYNC(qk_max[reuse_kv_idx], mask));
      }
      if (lane == 0) {
        red_smem[reuse_kv_idx][warp_idx] = qk_max[reuse_kv_idx];
      }
      __syncthreads();

      // TODO(woosuk): Refactor this part.
      // Get the max qk value for the sequence.
      qk_max[reuse_kv_idx] = lane < NUM_WARPS ? red_smem[reuse_kv_idx][lane] : -FLT_MAX;
    #pragma unroll
      for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
        qk_max[reuse_kv_idx] = fmaxf(qk_max[reuse_kv_idx], VLLM_SHFL_XOR_SYNC(qk_max[reuse_kv_idx], mask));
      }
      // Broadcast the max qk value to all threads.
      qk_max[reuse_kv_idx] = VLLM_SHFL_SYNC(qk_max[reuse_kv_idx], 0);

      for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
        float val = __expf(logits[(reuse_kv_idx * partition_size) + i] - qk_max[reuse_kv_idx]);
        logits[(reuse_kv_idx * partition_size) + i] = val;
        exp_sum[reuse_kv_idx] += val;
      }
      exp_sum[reuse_kv_idx] = block_sum<NUM_WARPS>(&red_smem[reuse_kv_idx][NUM_WARPS], exp_sum[reuse_kv_idx]);

      // Compute softmax.
      const float inv_sum = __fdividef(1.f, exp_sum[reuse_kv_idx] + 1e-6f);
      for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
        logits[(reuse_kv_idx * partition_size) + i] *= inv_sum;
      }
      __syncthreads();

      // If partitioning is enabled, store the max logit and exp_sum.
      if (USE_PARTITIONING && thread_idx == 0) {
        float* max_logits_ptr = max_logits +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions + partition_idx;
        *max_logits_ptr = qk_max[reuse_kv_idx];
        float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions +
                              head_idx * max_num_partitions + partition_idx;
        *exp_sums_ptr = exp_sum[reuse_kv_idx];
      }
    }
  }
  // Each thread will fetch 16 bytes from the value cache at a time.
  constexpr int V_VEC_SIZE = MIN(16 / sizeof(scalar_t), BLOCK_SIZE);
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using V_quant_vec = typename Vec<cache_t, V_VEC_SIZE>::Type;
  using Float_L_vec = typename FloatVec<L_vec>::Type;

  constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
  constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
  constexpr int NUM_ROWS_PER_THREAD =
      DIVIDE_ROUND_UP(HEAD_SIZE, NUM_ROWS_PER_ITER);

  // NOTE(woosuk): We use FP32 for the accumulator for better accuracy.
  float accs[REUSE_KV_TIMES][NUM_ROWS_PER_THREAD];

  #pragma unroll
  for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
    #pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        accs[reuse_kv_idx][i] = 0.f;
    }
  }
  scalar_t zero_value;
  zero(zero_value);
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx;
       block_idx += NUM_WARPS) {
    const int64_t physical_block_number =
        static_cast<int64_t>(block_table[block_idx]);
    const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
    const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
    L_vec logits_vec;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    V_vec v_vec;
    for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
      // NOTE(woosuk): The block number is stored in int32. However, we cast it to
      // int64 because int32 can lead to overflow when this variable is multiplied
      // by large numbers (e.g., kv_block_stride).
      // For blocksparse attention: skip computation on blocks that are not
      // attended
      // blocksparse specific vars
      const int head_idx = head_idx_soffset + reuse_kv_idx;
      int bs_block_offset;
      int q_bs_block_id;
      if constexpr (IS_BLOCK_SPARSE) {
        // const int num_blocksparse_blocks = DIVIDE_ROUND_UP(seq_len,
        // blocksparse_block_size);
        q_bs_block_id = (seq_len - 1) / blocksparse_block_size;
        if (blocksparse_head_sliding_step >= 0)
          // sliding on q heads
          bs_block_offset =
              (tp_rank * num_heads + head_idx) * blocksparse_head_sliding_step + 1;
        else
          // sliding on kv heads
          bs_block_offset = (tp_rank * num_kv_heads + kv_head_idx) *
                                (-blocksparse_head_sliding_step) +
                            1;
      }
      if constexpr (IS_BLOCK_SPARSE) {
        int v_bs_block_id = block_idx * BLOCK_SIZE / blocksparse_block_size;
        if (!((v_bs_block_id + bs_block_offset) % blocksparse_vert_stride == 0) &&
            !((v_bs_block_id > q_bs_block_id - blocksparse_local_blocks))) {
          continue;
        }
      }
      if(!odd_nheads || head_idx < q_boundary) {


      const cache_t* v_ptr = v_cache + physical_block_number * kv_block_stride
                                   + kv_head_idx * kv_head_stride;

     from_float(logits_vec, *reinterpret_cast<Float_L_vec*>(logits + (reuse_kv_idx * partition_size) +  token_idx - start_token_idx));
      // scalar_t* logits_vec_ptr = reinterpret_cast<scalar_t*>(&logits_vec);
      // for(int i=0;i<8;++i){
      //   from_float(*(logits_vec_ptr+i), 1000);
      // }

      if(reuse_kv_idx==0) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE) {
        const int offset = row_idx * BLOCK_SIZE + physical_block_offset;

        if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
        } else {
          V_quant_vec v_quant_vec =
              *reinterpret_cast<const V_quant_vec*>(v_ptr + offset);
          // Vector conversion from V_quant_vec to V_vec.
          v_vec = fp8::scaled_convert<V_vec, V_quant_vec, KV_DTYPE>(v_quant_vec,
                                                                    kv_scale);
        }
        if (block_idx == num_seq_blocks - 1) {
          // NOTE(woosuk): When v_vec contains the tokens that are out of the
          // context, we should explicitly zero out the values since they may
          // contain NaNs. See
          // https://github.com/vllm-project/vllm/issues/641#issuecomment-1682544472
          scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vec);
#pragma unroll
          for (int j = 0; j < V_VEC_SIZE; j++) {
            v_vec_ptr[j] = token_idx + j < seq_len ? v_vec_ptr[j] : zero_value;
          }
        }
        // if(threadIdx.x==0){
        //   scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vec);
        //   scalar_t* logits_vec_ptr = reinterpret_cast<scalar_t*>(&logits_vec);
        //   for(int i=0;i<8;++i){
        //     printf("v_vec[%d] = %f\n",i, half_to_float(v_vec_ptr[i]));
        //     // from_float(*(v_vec_ptr + i), 1000);
        //   }
        //   for(int i=0;i<8;++i){
        //     printf("logits_vec[%d] = %f\n",i,half_to_float(logits_vec_ptr[i]));
        //     // from_float(*(logits_vec_ptr + i), 1000);
        //   }
        // }
        // accs[reuse_kv_idx][i] += dot(logits_vec, v_vec);
      }
      } 
        accs[reuse_kv_idx][i] += dot(logits_vec, v_vec);
      }
      }
    }
  }

  // Perform reduction within each warp.
  #pragma unroll
  for(int reuse_kv_idx=0; reuse_kv_idx<REUSE_KV_TIMES; reuse_kv_idx++) {
    int head_idx = head_idx_soffset + reuse_kv_idx;

    if(!odd_nheads || head_idx < q_boundary) {
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    float acc = accs[reuse_kv_idx][i];
#pragma unroll
    for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
      acc += VLLM_SHFL_XOR_SYNC(acc, mask);
    }
    accs[reuse_kv_idx][i] = acc;
  }

  // NOTE(woosuk): A barrier is required because the shared memory space for
  // logits is reused for the output.
  __syncthreads();

  // Perform reduction across warps.
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    // Upper warps write to shared memory.
    if (warp_idx >= mid && warp_idx < i) {
       float* dst = &out_smem[(reuse_kv_idx * (NUM_WARPS / 2) * HEAD_SIZE) + (warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[reuse_kv_idx][i];
        }
      }
    }
    __syncthreads();

    // Lower warps update the output.
    if (warp_idx < mid) {
      const float* src = &out_smem[(reuse_kv_idx * (NUM_WARPS / 2) * HEAD_SIZE) + warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[reuse_kv_idx][i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }

  // Write the final output.
  if (warp_idx == 0) {
    scalar_t* out_ptr =
        out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE + partition_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        from_float(*(out_ptr + row_idx), accs[reuse_kv_idx][i]);
      }
    }
  }
  }
  }
}


// Grid: (num_heads, num_seqs, 1).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          int REUSE_KV_TIMES = 1,
          bool IS_BLOCK_SPARSE,
          bool odd_nheads = false>
__global__ __launch_bounds__(256,1) void paged_attention_v1_kernel(
    scalar_t* __restrict__ out,           // [num_seqs, num_heads, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_heads,               // [num_heads]    
    const int num_kv_heads,               // [num_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
      paged_attention_kernel<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                          KV_DTYPE, IS_BLOCK_SPARSE, REUSE_KV_TIMES, odd_nheads>(
        /* exp_sums */ nullptr, /* max_logits */ nullptr, out, q, k_cache,
        v_cache, num_heads, num_kv_heads, scale, block_tables, seq_lens,
        max_num_blocks_per_seq, alibi_slopes, q_stride, kv_block_stride,
        kv_head_stride, kv_scale, tp_rank, blocksparse_local_blocks,
        blocksparse_vert_stride, blocksparse_block_size,
        blocksparse_head_sliding_step);
}

// Grid: (num_heads, num_seqs, max_num_partitions).
template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int REUSE_KV_TIMES,
          int PARTITION_SIZE,
          bool odd_nheads = false>
__global__ __launch_bounds__(256,1) void paged_attention_v2_kernel(
    float* __restrict__ exp_sums,  // [num_seqs, num_heads, max_num_partitions]
    float* __restrict__ max_logits,       // [num_seqs, num_heads,
                                          // max_num_partitions]
    scalar_t* __restrict__ tmp_out,       // [num_seqs, num_heads,
                                          // max_num_partitions, head_size]
    const scalar_t* __restrict__ q,       // [num_seqs, num_heads, head_size]
    const cache_t* __restrict__ k_cache,  // [num_blocks, num_kv_heads,
                                          // head_size/x, block_size, x]
    const cache_t* __restrict__ v_cache,  // [num_blocks, num_kv_heads,
                                          // head_size, block_size]
    const int num_heads,               // [num_heads]                                      
    const int num_kv_heads,               // [num_kv_heads]
    const float scale,
    const int* __restrict__ block_tables,  // [num_seqs, max_num_blocks_per_seq]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,  // [num_heads]
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float kv_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
        paged_attention_kernel<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE, REUSE_KV_TIMES, odd_nheads, PARTITION_SIZE>(
          exp_sums, max_logits, tmp_out, q, k_cache, v_cache, num_heads, num_kv_heads, scale,
          block_tables, seq_lens, max_num_blocks_per_seq, alibi_slopes, q_stride,
          kv_block_stride, kv_head_stride, kv_scale, tp_rank,
          blocksparse_local_blocks, blocksparse_vert_stride, blocksparse_block_size,
          blocksparse_head_sliding_step);
}

// Grid: (num_heads, num_seqs).
template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE>
__global__ __launch_bounds__(256,1) void paged_attention_v2_reduce_kernel(
    scalar_t* __restrict__ out,            // [num_seqs, num_heads, head_size]
    const float* __restrict__ exp_sums,    // [num_seqs, num_heads,
                                           // max_num_partitions]
    const float* __restrict__ max_logits,  // [num_seqs, num_heads,
                                           // max_num_partitions]
    const scalar_t* __restrict__ tmp_out,  // [num_seqs, num_heads,
                                           // max_num_partitions, head_size]
    const int* __restrict__ seq_lens,      // [num_seqs]
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int seq_len = seq_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(seq_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    // No need to reduce. Only copy tmp_out to out.
    scalar_t* out_ptr =
        out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    const scalar_t* tmp_out_ptr =
        tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE;
    for (int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
      out_ptr[i] = tmp_out_ptr[i];
    }
    // Terminate the thread block.
    return;
  }

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warp_idx = threadIdx.x / WARP_SIZE;
  const int lane = threadIdx.x % WARP_SIZE;

  // Size: 2 * num_partitions.
  extern __shared__ char shared_mem[];
  // Workspace for reduction.
  __shared__ float red_smem[2 * NUM_WARPS];

  // Load max logits to shared memory.
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  const float* max_logits_ptr = max_logits +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions;
  float max_logit = -FLT_MAX;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    const float l = max_logits_ptr[i];
    shared_max_logits[i] = l;
    max_logit = fmaxf(max_logit, l);
  }
  __syncthreads();

  // Get the global max logit.
  // Reduce within the warp.
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = max_logit;
  }
  __syncthreads();
  // Reduce across warps.
  max_logit = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  // Broadcast the max value to all threads.
  max_logit = VLLM_SHFL_SYNC(max_logit, 0);

  // Load rescaled exp sums to shared memory.
  float* shared_exp_sums =
      reinterpret_cast<float*>(shared_mem + sizeof(float) * num_partitions);
  const float* exp_sums_ptr = exp_sums +
                              seq_idx * num_heads * max_num_partitions +
                              head_idx * max_num_partitions;
  float global_exp_sum = 0.0f;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    float l = shared_max_logits[i];
    float rescaled_exp_sum = exp_sums_ptr[i] * expf(l - max_logit);
    global_exp_sum += rescaled_exp_sum;
    shared_exp_sums[i] = rescaled_exp_sum;
  }
  __syncthreads();
  global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.0f, global_exp_sum + 1e-6f);

  // Aggregate tmp_out to out.
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
      head_idx * max_num_partitions * HEAD_SIZE;
  scalar_t* out_ptr =
      out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
#pragma unroll
  for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
    float acc = 0.0f;
    for (int j = 0; j < num_partitions; ++j) {
      acc += to_float(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] *
             inv_global_exp_sum;
    }
    from_float(out_ptr[i], acc);
  }
}

}  // namespace vllm

#define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, REUSE_KV_TIMES, IS_BLOCK_SPARSE, odd_nheads>),  \
      shared_mem_size);                                                     \
 hipLaunchKernelGGL(( vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, REUSE_KV_TIMES, IS_BLOCK_SPARSE, odd_nheads>)   \
      , dim3(grid), dim3(block), shared_mem_size, stream,                            \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_heads, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          kv_scale, tp_rank, blocksparse_local_blocks,                      \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step);

// #define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                \
// vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
//                                   NUM_THREADS, KV_DTYPE, REUSE_KV_TIMES, IS_BLOCK_SPARSE, odd_nheads>   \
//       <<<dim3(grid), dim3(block)>>>(                           \
//           out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_heads, num_kv_heads, \
//           scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
//           alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
//           kv_scale, tp_rank, blocksparse_local_blocks,                      \
//           blocksparse_vert_stride, blocksparse_block_size,                  \
//           blocksparse_head_sliding_step);

// TODO(woosuk): Tune NUM_THREADS.
template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE>
void paged_attention_v1_launcher(
    torch::Tensor& out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, float kv_scale,
    const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);
  int num_threads = 128;
  if(num_heads!=num_kv_heads){
    num_threads =256;
  }
  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();

  int padded_max_seq_len = DIVIDE_ROUND_UP(max_seq_len, BLOCK_SIZE) * BLOCK_SIZE;
  REUSEKV_SWITCH_V1(num_heads * num_seqs , [&] {
    BOOL_SWITCH((num_heads/num_kv_heads % REUSE_KV_TIMES != 0), odd_nheads, [&] {
      HEADSIZE_SWITCH(head_size, [&] {
        NUM_THREADS_SWITCH(num_threads, [&] {
          OPT_SWITCH(num_heads == num_kv_heads, [&] {
          constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
          int logits_size =  REUSE_KV_TIMES*padded_max_seq_len * sizeof(float);
          int outputs_size =  REUSE_KV_TIMES*(NUM_WARPS / 2) * head_size * sizeof(float);
          // Python-side check in vllm.worker.worker._check_if_can_support_max_seq_len
          // Keep that in sync with the logic here!
          int shared_mem_size = ::max(logits_size, outputs_size);
          if(num_heads == num_kv_heads) shared_mem_size = ::max(12 * 1024, shared_mem_size);
          // int shared_mem_size = ::max(31*1024, ::max(logits_size, outputs_size));
          // std::cout<<"shared_mem_size = "<<shared_mem_size<<std::endl;
          dim3 grid((num_heads/num_kv_heads + REUSE_KV_TIMES - 1) / REUSE_KV_TIMES*num_kv_heads, 1, num_seqs);
          dim3 block(NUM_THREADS);
          const at::hip::OptionalHIPGuardMasqueradingAsCUDA device_guard(device_of(query));
          const hipStream_t stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();
          LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE);
          });
        });
      });
    });
  }); 
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)  \
  paged_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,              \
                              IS_BLOCK_SPARSE>(                              \
      out, query, key_cache, value_cache, num_kv_heads, scale, block_tables, \
      seq_lens, max_seq_len, alibi_slopes, kv_scale, tp_rank,                \
      blocksparse_local_blocks, blocksparse_vert_stride,                     \
      blocksparse_block_size, blocksparse_head_sliding_step);

#define CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  switch (is_block_sparse) {                                               \
    case true:                                                             \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);     \
      break;                                                               \
    case false:                                                            \
      CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);    \
      break;                                                               \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 16:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

// // NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// // 1, 2, 4, 64, 128, 256.
// #define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
//   switch (block_size) {                                           \
//     case 16:                                                      \
//       CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
//       break;                                                      \
//       TORCH_CHECK(false, "Unsupported block size: ", block_size); \
//       break;                                                      \
//   }

void paged_attention_v1(
    torch::Tensor& out,    // [num_seqs, num_heads, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double kv_scale, const int64_t tp_rank,
    const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);

  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V1_LAUNCHER_BLOCK_SIZE)
}

#define LAUNCH_PAGED_ATTENTION_V2(HEAD_SIZE)                                   \
 hipLaunchKernelGGL(( vllm::paged_attention_v2_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,           \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE,      \
                                  REUSE_KV_TIMES, PARTITION_SIZE, odd_nheads>)                              \
      , dim3(grid), dim3(block), shared_mem_size, stream,                               \
          exp_sums_ptr, max_logits_ptr, tmp_out_ptr, query_ptr, key_cache_ptr, \
          value_cache_ptr, num_heads, num_kv_heads, scale, block_tables_ptr,              \
          seq_lens_ptr, max_num_blocks_per_seq, alibi_slopes_ptr, q_stride,    \
          kv_block_stride, kv_head_stride, kv_scale, tp_rank,                  \
          blocksparse_local_blocks, blocksparse_vert_stride,                   \
          blocksparse_block_size, blocksparse_head_sliding_step);              \
 hipLaunchKernelGGL(( vllm::paged_attention_v2_reduce_kernel<T, HEAD_SIZE, NUM_THREADS,            \
                                         PARTITION_SIZE>)                       \
      , dim3(reduce_grid), dim3(block), reduce_shared_mem_size, stream,                 \
          out_ptr, exp_sums_ptr, max_logits_ptr, tmp_out_ptr, seq_lens_ptr,    \
          max_num_partitions);

template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE,
          int NUM_THREADS = 256, int PARTITION_SIZE = 512>
void paged_attention_v2_launcher(
    torch::Tensor& out, torch::Tensor& exp_sums, torch::Tensor& max_logits,
    torch::Tensor& tmp_out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, float kv_scale,
    const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);
  int kv_block_stride = key_cache.stride(0);
  int kv_head_stride = key_cache.stride(1);

  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr =
      alibi_slopes
          ? reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
          : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  float* exp_sums_ptr = reinterpret_cast<float*>(exp_sums.data_ptr());
  float* max_logits_ptr = reinterpret_cast<float*>(max_logits.data_ptr());
  T* tmp_out_ptr = reinterpret_cast<T*>(tmp_out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int max_num_partitions = DIVIDE_ROUND_UP(max_seq_len, PARTITION_SIZE);
  REUSEKV_SWITCH(num_heads * max_num_partitions * num_seqs , [&] {
    BOOL_SWITCH((num_heads/num_kv_heads % REUSE_KV_TIMES != 0), odd_nheads, [&] {
      HEADSIZE_SWITCH(head_size, [&] {
        OPT_SWITCH(num_heads == num_kv_heads, [&] {
        int logits_size = REUSE_KV_TIMES*PARTITION_SIZE * sizeof(float);
        int outputs_size = REUSE_KV_TIMES*(NUM_WARPS / 2) * head_size * sizeof(float);

        // For paged attention v2 kernel.
        // dim3 grid(num_heads, max_num_partitions, num_seqs);

        dim3 grid;
        grid.x = (num_heads/num_kv_heads + REUSE_KV_TIMES -1)/REUSE_KV_TIMES * num_kv_heads;
        grid.y = max_num_partitions;
        grid.z = num_seqs;
        // int shared_mem_size = ::max(1024*32, ::max(logits_size, outputs_size));
        int shared_mem_size = ::max(logits_size, outputs_size);
        // For paged attention v2 reduce kernel.
        dim3 reduce_grid(num_heads, num_seqs);
        int reduce_shared_mem_size = 2 * max_num_partitions * sizeof(float);
        dim3 block(NUM_THREADS);
        const at::hip::OptionalHIPGuardMasqueradingAsCUDA device_guard(device_of(query));
        const hipStream_t stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();
        LAUNCH_PAGED_ATTENTION_V2(HEAD_SIZE);
        });
      });
    });
  });
}

#define CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)   \
  paged_attention_v2_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,               \
                              IS_BLOCK_SPARSE>(                               \
      out, exp_sums, max_logits, tmp_out, query, key_cache, value_cache,      \
      num_kv_heads, scale, block_tables, seq_lens, max_seq_len, alibi_slopes, \
      kv_scale, tp_rank, blocksparse_local_blocks, blocksparse_vert_stride,   \
      blocksparse_block_size, blocksparse_head_sliding_step);

#define CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  switch (is_block_sparse) {                                               \
    case true:                                                             \
      CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);     \
      break;                                                               \
    case false:                                                            \
      CALL_V2_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);    \
      break;                                                               \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V2_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 16:                                                      \
      CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

// // NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// // 1, 2, 4, 64, 128, 256.
// #define CALL_V2_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
//   switch (block_size) {                                           \
//     case 16:                                                      \
//       CALL_V2_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
//       break;                                                      \
//       TORCH_CHECK(false, "Unsupported block size: ", block_size); \
//       break;                                                      \
//   }

void paged_attention_v2(
    torch::Tensor& out,         // [num_seqs, num_heads, head_size]
    torch::Tensor& exp_sums,    // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor& max_logits,  // [num_seqs, num_heads, max_num_partitions]
    torch::Tensor&
        tmp_out,  // [num_seqs, num_heads, max_num_partitions, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, double kv_scale, const int64_t tp_rank,
    const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);
  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V2_LAUNCHER_BLOCK_SIZE)
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
